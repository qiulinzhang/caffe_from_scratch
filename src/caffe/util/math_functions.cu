#include "hip/hip_runtime.h"
#include <math_functions.h> // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/rng.hpp"

namespace caffe {
    template <>
    void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB,
                               const int M, const int N, const int K,
                               const float alpha, const float* A, const float* B,
                               const float beta, float* C) {
        // Note that cublas follows fortran order
        int lda = (TransA == CblasNoTrans) ? K : M;
        int ldb = (TransB == CblasNoTrans) ? N : K;
        hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
        hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
        CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransA, cuTransB, 
                     N, M, K,
                     &alpha, B, ldb, A, lda,
                     &beta, C, N));
    }
}